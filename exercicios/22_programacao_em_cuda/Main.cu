
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <float.h>

#define Lx 1.0
#define Ly 1.0
#define Lz 1.0
#define Nx 241  // Igual em ambos!
#define Ny 241  // Igual em ambos!
#define Nz 241  // Igual em ambos!
#define NU 0.1
#define DT 0.00002
#define T 0.01
#define SIGMA 0.1

// Macro para verificação de erros CUDA
#define CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "Erro CUDA (arquivo: %s, linha: %d): %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

double elapsed(struct timeval t0, struct timeval t1) {
    return (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec) / 1e6;
}

// Kernel para zerar as bordas
__global__ void set_borders_zero(double* u_new, int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Face X = 0 e X = nx-1
    if (i < ny && j < nz) {
        u_new[0 * ny * nz + i * nz + j] = 0.0;
        u_new[(nx-1) * ny * nz + i * nz + j] = 0.0;
    }

    // Face Y = 0 e Y = ny-1
    if (i < nx && j < nz) {
        u_new[i * ny * nz + 0 * nz + j] = 0.0;
        u_new[i * ny * nz + (ny-1) * nz + j] = 0.0;
    }

    // Face Z = 0 e Z = nz-1
    if (i < nx && j < ny) {
        u_new[i * ny * nz + j * nz + 0] = 0.0;
        u_new[i * ny * nz + j * nz + (nz-1)] = 0.0;
    }
}

// Kernel principal de evolução
__global__ void evolve_kernel(double* u, double* u_new, 
                             double nu_dt, double dx2, double dy2, double dz2,
                             int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i < nx - 1 && j < ny - 1 && k < nz - 1) {
        int idx = i * ny * nz + j * nz + k;
        
        double u_ijk = u[idx];
        double d2x = (u[(i+1)*ny*nz + j*nz + k] - 2*u_ijk + u[(i-1)*ny*nz + j*nz + k]) / dx2;
        double d2y = (u[i*ny*nz + (j+1)*nz + k] - 2*u_ijk + u[i*ny*nz + (j-1)*nz + k]) / dy2;
        double d2z = (u[i*ny*nz + j*nz + (k+1)] - 2*u_ijk + u[i*ny*nz + j*nz + (k-1)]) / dz2;
        
        u_new[idx] = u_ijk + nu_dt * (d2x + d2y + d2z);
    }
}

// Kernel para redução de estatísticas usando memória compartilhada
__global__ void reduce_stats_kernel(double* u, double* stats, int size) {
    extern __shared__ double sdata[];
    
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    // Inicializa valores locais
    double my_sum = 0.0;
    double my_max = -DBL_MAX;
    double my_min = DBL_MAX;
    double my_sum_sq = 0.0;
    
    // Carrega até 4 elementos por thread
    if (i < size) {
        my_sum = u[i];
        my_max = u[i];
        my_min = u[i];
        my_sum_sq = u[i] * u[i];
    }
    if (i + blockDim.x < size) {
        double val = u[i + blockDim.x];
        my_sum += val;
        my_max = fmax(my_max, val);
        my_min = fmin(my_min, val);
        my_sum_sq += val * val;
    }
    if (i + 2*blockDim.x < size) {
        double val = u[i + 2*blockDim.x];
        my_sum += val;
        my_max = fmax(my_max, val);
        my_min = fmin(my_min, val);
        my_sum_sq += val * val;
    }
    if (i + 3*blockDim.x < size) {
        double val = u[i + 3*blockDim.x];
        my_sum += val;
        my_max = fmax(my_max, val);
        my_min = fmin(my_min, val);
        my_sum_sq += val * val;
    }
    
    // Memória compartilhada para 4 valores por thread (soma, max, min, sum_sq)
    int idx = tid * 4;
    sdata[idx] = my_sum;
    sdata[idx+1] = my_max;
    sdata[idx+2] = my_min;
    sdata[idx+3] = my_sum_sq;
    __syncthreads();
    
    // Redução em árvore
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            int sidx = tid * 4;
            int sidx2 = (tid + s) * 4;
            
            sdata[sidx] += sdata[sidx2];       // Soma
            sdata[sidx+1] = fmax(sdata[sidx+1], sdata[sidx2+1]); // Máximo
            sdata[sidx+2] = fmin(sdata[sidx+2], sdata[sidx2+2]); // Mínimo
            sdata[sidx+3] += sdata[sidx2+3];   // Soma dos quadrados
        }
        __syncthreads();
    }
    
    // Thread 0 escreve o resultado do bloco
    if (tid == 0) {
        int bidx = blockIdx.x * 4;
        stats[bidx] = sdata[0];       // Soma
        stats[bidx+1] = sdata[1];     // Máximo
        stats[bidx+2] = sdata[2];     // Mínimo
        stats[bidx+3] = sdata[3];     // Soma dos quadrados
    }
}

// Função wrapper para cálculo de estatísticas na GPU
void compute_stats_gpu(double* d_u, double* h_stats, int size) {
    int block_size = 256;
    int grid_size = (size + 4 * block_size - 1) / (4 * block_size);
    
    double* d_stats;
    CHECK(hipMalloc(&d_stats, grid_size * 4 * sizeof(double)));
    
    size_t shared_mem_size = block_size * 4 * sizeof(double);
    reduce_stats_kernel<<<grid_size, block_size, shared_mem_size>>>(d_u, d_stats, size);
    CHECK(hipGetLastError());
    
    double* block_stats = (double*)malloc(grid_size * 4 * sizeof(double));
    CHECK(hipMemcpy(block_stats, d_stats, grid_size * 4 * sizeof(double), hipMemcpyDeviceToHost));
    
    // Redução final na CPU (pequena)
    double total_sum = 0.0;
    double global_max = -DBL_MAX;
    double global_min = DBL_MAX;
    double total_sum_sq = 0.0;
    
    for (int i = 0; i < grid_size; i++) {
        int idx = i * 4;
        total_sum += block_stats[idx];
        global_max = fmax(global_max, block_stats[idx+1]);
        global_min = fmin(global_min, block_stats[idx+2]);
        total_sum_sq += block_stats[idx+3];
    }
    
    h_stats[0] = total_sum;
    h_stats[1] = global_max;
    h_stats[2] = global_min;
    h_stats[3] = total_sum_sq;
    
    free(block_stats);
    CHECK(hipFree(d_stats));
}

void print_stats(double mass, double max_val, double min_val, double l2_squared, int step) {
    printf("Passo %d:\n", step);
    printf("  Massa total: %.6f\n", mass);
    printf("  Valor máximo: %.6f\n", max_val);
    printf("  Valor mínimo: %.6f\n", min_val);
    printf("  Norma L2: %.6f\n\n", sqrt(l2_squared));
}

int main() {
    int nt = (int)(T / DT);
    double dx = Lx / (Nx - 1);
    double dy = Ly / (Ny - 1);
    double dz = Lz / (Nz - 1);
    double dx2 = dx * dx;
    double dy2 = dy * dy;
    double dz2 = dz * dz;
    double nu_dt = NU * DT;
    int total_size = Nx * Ny * Nz;

    double *u = (double*)malloc(total_size * sizeof(double));
    double *u_new = (double*)malloc(total_size * sizeof(double));
    double *x = (double*)malloc(Nx * sizeof(double));
    double *y = (double*)malloc(Ny * sizeof(double));
    double *z = (double*)malloc(Nz * sizeof(double));
    int i, j, k, n;

    // Inicialização
    for (i = 0; i < Nx; i++) x[i] = i * dx;
    for (j = 0; j < Ny; j++) y[j] = j * dy;
    for (k = 0; k < Nz; k++) z[k] = k * dz;

    // Inicialização com zeros
    memset(u, 0, total_size * sizeof(double));
    memset(u_new, 0, total_size * sizeof(double));

    // Condição inicial gaussiana
    double cx = Lx/2, cy = Ly/2, cz = Lz/2;
    for (i = 0; i < Nx; i++) {
        for (j = 0; j < Ny; j++) {
            for (k = 0; k < Nz; k++) {
                double dist2 = pow(x[i] - cx, 2) + pow(y[j] - cy, 2) + pow(z[k] - cz, 2);
                u[i*Ny*Nz + j*Nz + k] = exp(-dist2 / (2 * SIGMA * SIGMA));
            }
        }
    }

    // Alocação na GPU
    double *d_u, *d_u_new;
    CHECK(hipMalloc(&d_u, total_size * sizeof(double)));
    CHECK(hipMalloc(&d_u_new, total_size * sizeof(double)));
    
    // Copiar dados iniciais para GPU
    CHECK(hipMemcpy(d_u, u, total_size * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_u_new, u_new, total_size * sizeof(double), hipMemcpyHostToDevice));

    // Configuração de kernels
    dim3 block_borders(16, 16);
    int max_dim1 = (Nx > Ny) ? Nx : Ny;
    int max_dim2 = (Ny > Nz) ? Ny : Nz;
    if (Nz > max_dim2) max_dim2 = Nz;
    dim3 grid_borders(
        (max_dim1 + block_borders.x - 1) / block_borders.x,
        (max_dim2 + block_borders.y - 1) / block_borders.y
    );
    
    dim3 block_evolve(8, 8, 4);
    dim3 grid_evolve(
        (Nx - 2 + block_evolve.x - 1) / block_evolve.x,
        (Ny - 2 + block_evolve.y - 1) / block_evolve.y,
        (Nz - 2 + block_evolve.z - 1) / block_evolve.z
    );

    // Variáveis para estatísticas
    double stats[4];
    
    // Cálculo das estatísticas iniciais na GPU
    compute_stats_gpu(d_u, stats, total_size);
    print_stats(stats[0], stats[1], stats[2], stats[3], 0);

    struct timeval t0, t1;
    gettimeofday(&t0, NULL);

    // Loop de evolução temporal
    for (n = 0; n < nt; n++) {
        // Zerar bordas na GPU
        set_borders_zero<<<grid_borders, block_borders>>>(d_u_new, Nx, Ny, Nz);
        CHECK(hipGetLastError());

        // Calcular evolução
        evolve_kernel<<<grid_evolve, block_evolve>>>(d_u, d_u_new, nu_dt, dx2, dy2, dz2, Nx, Ny, Nz);
        CHECK(hipGetLastError());

        // Trocar ponteiros
        double* temp = d_u;
        d_u = d_u_new;
        d_u_new = temp;

        // Calcular estatísticas periodicamente na GPU
        if ((n+1) % (nt/10) == 0 || n == nt-1) {
            compute_stats_gpu(d_u, stats, total_size);
            print_stats(stats[0], stats[1], stats[2], stats[3], n+1);
        }
    }

    gettimeofday(&t1, NULL);

    // Limpeza
    free(u);
    free(u_new);
    free(x);
    free(y);
    free(z);
    CHECK(hipFree(d_u));
    CHECK(hipFree(d_u_new));

    printf("Simulação 3D concluída (CUDA)\n");
    printf("Tempo de execução: %.6lf segundos\n", elapsed(t0, t1));
    printf("Nx = %d, Ny = %d, Nz = %d\n", Nx, Ny, Nz);
    
    return 0;
}