
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define Lx 1.0
#define Ly 1.0
#define Lz 1.0
#define Nx 181
#define Ny 181
#define Nz 181
#define NU 0.01
#define DT 0.0005
#define T 0.1
#define SIGMA 0.1

// Macro para verificação de erros CUDA
#define CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "Erro CUDA (arquivo: %s, linha: %d): %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

double elapsed(struct timeval t0, struct timeval t1) {
    return (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec) / 1e6;
}

// Kernel para zerar as bordas
__global__ void set_borders_zero(double* u, int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Face x = 0
    if (i < ny && j < nz) {
        u[0 * ny * nz + i * nz + j] = 0.0;
    }
    
    // Face x = nx-1
    if (i < ny && j < nz) {
        u[(nx-1) * ny * nz + i * nz + j] = 0.0;
    }
    
    // Face y = 0
    if (i < nx && j < nz) {
        u[i * ny * nz + 0 * nz + j] = 0.0;
    }
    
    // Face y = ny-1
    if (i < nx && j < nz) {
        u[i * ny * nz + (ny-1) * nz + j] = 0.0;
    }
    
    // Face z = 0
    if (i < nx && j < ny) {
        u[i * ny * nz + j * nz + 0] = 0.0;
    }
    
    // Face z = nz-1
    if (i < nx && j < ny) {
        u[i * ny * nz + j * nz + (nz-1)] = 0.0;
    }
}

// Kernel principal de evolução
__global__ void evolve_kernel(double* u, double* u_new, 
                              double nu_dt, double dx2, double dy2, double dz2,
                              int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i < nx - 1 && j < ny - 1 && k < nz - 1) {
        int idx = i * ny * nz + j * nz + k;
        
        double u_ijk = u[idx];
        double d2x = (u[(i+1)*ny*nz + j*nz + k] - 2*u_ijk + u[(i-1)*ny*nz + j*nz + k]) / dx2;
        double d2y = (u[i*ny*nz + (j+1)*nz + k] - 2*u_ijk + u[i*ny*nz + (j-1)*nz + k]) / dy2;
        double d2z = (u[i*ny*nz + j*nz + (k+1)] - 2*u_ijk + u[i*ny*nz + j*nz + (k-1)]) / dz2;
        
        u_new[idx] = u_ijk + nu_dt * (d2x + d2y + d2z);
    }
}

int main() {
    int nt = (int)(T / DT);
    double dx = Lx / (Nx - 1);
    double dy = Ly / (Ny - 1);
    double dz = Lz / (Nz - 1);
    double dx2 = dx * dx;
    double dy2 = dy * dy;
    double dz2 = dz * dz;
    double nu_dt = NU * DT;

    double *u = (double*)malloc(Nx * Ny * Nz * sizeof(double));
    double *u_new = (double*)malloc(Nx * Ny * Nz * sizeof(double));
    double *x = (double*)malloc(Nx * sizeof(double));
    double *y = (double*)malloc(Ny * sizeof(double));
    double *z = (double*)malloc(Nz * sizeof(double));
    int i, j, k, n;

    for (i = 0; i < Nx; i++) x[i] = i * dx;
    for (j = 0; j < Ny; j++) y[j] = j * dy;
    for (k = 0; k < Nz; k++) z[k] = k * dz;

    // Inicialização
    for (i = 0; i < Nx; i++)
        for (j = 0; j < Ny; j++)
            for (k = 0; k < Nz; k++)
                u[i*Ny*Nz + j*Nz + k] = 0.0;

    FILE *fp = fopen("evolucao3d.csv", "w");
    if (!fp) {
        printf("Erro ao abrir arquivo!\n");
        return 1;
    }
    for (i = 0; i < Nx; i++)
        for (j = 0; j < Ny; j++)
            for (k = 0; k < Nz; k++)
                fprintf(fp, "%lf%c", u[i*Ny*Nz + j*Nz + k], (k < Nz-1) ? ',' : '\n');
    fprintf(fp, "\n");

    // Condição inicial gaussiana
    double cx = Lx/2, cy = Ly/2, cz = Lz/2;
    for (i = 0; i < Nx; i++)
        for (j = 0; j < Ny; j++)
            for (k = 0; k < Nz; k++) {
                double dist2 = pow(x[i] - cx, 2) + pow(y[j] - cy, 2) + pow(z[k] - cz, 2);
                u[i*Ny*Nz + j*Nz + k] = exp(-dist2 / (2 * SIGMA * SIGMA));
            }

    for (i = 0; i < Nx; i++)
        for (j = 0; j < Ny; j++)
            for (k = 0; k < Nz; k++)
                fprintf(fp, "%lf%c", u[i*Ny*Nz + j*Nz + k], (k < Nz-1) ? ',' : '\n');
    fprintf(fp, "\n");

    // Alocação na GPU
    double *d_u, *d_u_new;
    CHECK(hipMalloc(&d_u, Nx * Ny * Nz * sizeof(double)));
    CHECK(hipMalloc(&d_u_new, Nx * Ny * Nz * sizeof(double)));
    
    // Copiar dados iniciais para GPU
    CHECK(hipMemcpy(d_u, u, Nx * Ny * Nz * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_u_new, u_new, Nx * Ny * Nz * sizeof(double), hipMemcpyHostToDevice));

    // Configuração de kernels
    dim3 block_borders(16, 16);
    dim3 grid_borders((Ny + 15) / 16, (Nz + 15) / 16);
    
    dim3 block_evolve(8, 8, 4);
    dim3 grid_evolve(
        (Nx - 2 + block_evolve.x - 1) / block_evolve.x,
        (Ny - 2 + block_evolve.y - 1) / block_evolve.y,
        (Nz - 2 + block_evolve.z - 1) / block_evolve.z
    );

    struct timeval t0, t1;
    gettimeofday(&t0, NULL);

    for (n = 0; n < nt; n++) {
        // Zerar bordas na GPU
        set_borders_zero<<<grid_borders, block_borders>>>(d_u_new, Nx, Ny, Nz);
        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());

        // Calcular evolução
        evolve_kernel<<<grid_evolve, block_evolve>>>(d_u, d_u_new, nu_dt, dx2, dy2, dz2, Nx, Ny, Nz);
        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());

        // Trocar ponteiros
        double* temp = d_u;
        d_u = d_u_new;
        d_u_new = temp;

        // Escrever resultados periodicamente
        if ((n+1) % (nt/4) == 0 || n == nt-1) {
            CHECK(hipMemcpy(u, d_u, Nx * Ny * Nz * sizeof(double), hipMemcpyDeviceToHost));
            for (i = 0; i < Nx; i++)
                for (j = 0; j < Ny; j++)
                    for (k = 0; k < Nz; k++)
                        fprintf(fp, "%lf%c", u[i*Ny*Nz + j*Nz + k], (k < Nz-1) ? ',' : '\n');
            fprintf(fp, "\n");
        }
    }

    gettimeofday(&t1, NULL);

    fclose(fp);
    free(u);
    free(u_new);
    free(x);
    free(y);
    free(z);
    CHECK(hipFree(d_u));
    CHECK(hipFree(d_u_new));

    printf("Simulação 3D concluída (CUDA). Resultados salvos em evolucao3d.csv\n");
    printf("Tempo de execução: %.6lf segundos\n", elapsed(t0, t1));
    printf("Nx = %d, Ny = %d, Nz = %d\n", Nx, Ny, Nz);
    
    return 0;
}