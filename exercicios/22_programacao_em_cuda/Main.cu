
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <float.h>

#define Lx 1.0
#define Ly 1.0
#define Lz 1.0
#define Nx 301
#define Ny 301
#define Nz 301
#define NU 0.1
#define DT 0.00002
#define T 0.01
#define SIGMA 0.1

#define CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "Erro CUDA (arquivo: %s, linha: %d): %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

double elapsed(struct timeval t0, struct timeval t1) {
    return (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec) / 1e6;
}

__global__ void set_borders_zero(double* u_new, int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < ny && j < nz) {
        u_new[0 * ny * nz + i * nz + j] = 0.0;
        u_new[(nx-1) * ny * nz + i * nz + j] = 0.0;
    }

    if (i < nx && j < nz) {
        u_new[i * ny * nz + 0 * nz + j] = 0.0;
        u_new[i * ny * nz + (ny-1) * nz + j] = 0.0;
    }

    if (i < nx && j < ny) {
        u_new[i * ny * nz + j * nz + 0] = 0.0;
        u_new[i * ny * nz + j * nz + (nz-1)] = 0.0;
    }
}

__global__ void evolve_kernel(double* u, double* u_new, 
                             double nu_dt, double dx2, double dy2, double dz2,
                             int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i < nx - 1 && j < ny - 1 && k < nz - 1) {
        int idx = i * ny * nz + j * nz + k;
        
        double u_ijk = u[idx];
        double d2x = (u[(i+1)*ny*nz + j*nz + k] - 2*u_ijk + u[(i-1)*ny*nz + j*nz + k]) / dx2;
        double d2y = (u[i*ny*nz + (j+1)*nz + k] - 2*u_ijk + u[i*ny*nz + (j-1)*nz + k]) / dy2;
        double d2z = (u[i*ny*nz + j*nz + (k+1)] - 2*u_ijk + u[i*ny*nz + j*nz + (k-1)]) / dz2;
        
        u_new[idx] = u_ijk + nu_dt * (d2x + d2y + d2z);
    }
}

// Kernel com redução simplificada (sem shared memory)
__global__ void partial_stats_kernel(double* u, double* partial_sums, double* partial_maxs, 
                                    double* partial_mins, double* partial_sum_sqs, int size, int partial_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    double local_sum = 0.0;
    double local_max = -DBL_MAX;
    double local_min = DBL_MAX;
    double local_sum_sq = 0.0;

    for (int i = idx; i < size; i += stride) {
        double val = u[i];
        local_sum += val;
        local_max = fmax(local_max, val);
        local_min = fmin(local_min, val);
        local_sum_sq += val * val;
    }

    // Apenas threads dentro do limite de partial_size escrevem
    if (idx < partial_size) {
        partial_sums[idx] = local_sum;
        partial_maxs[idx] = local_max;
        partial_mins[idx] = local_min;
        partial_sum_sqs[idx] = local_sum_sq;
    }
}

void compute_stats_gpu(double* d_u, double* h_stats, int size) {
    int block_size = 256;
    int grid_size = (size + block_size - 1) / block_size;
    if (grid_size > 1024) grid_size = 1024;

    // Tamanho real do array parcial
    int partial_size = grid_size * block_size;

    double *d_partial_sums, *d_partial_maxs, *d_partial_mins, *d_partial_sum_sqs;
    CHECK(hipMalloc(&d_partial_sums, partial_size * sizeof(double)));
    CHECK(hipMalloc(&d_partial_maxs, partial_size * sizeof(double)));
    CHECK(hipMalloc(&d_partial_mins, partial_size * sizeof(double)));
    CHECK(hipMalloc(&d_partial_sum_sqs, partial_size * sizeof(double)));

    partial_stats_kernel<<<grid_size, block_size>>>(d_u, d_partial_sums, d_partial_maxs, 
                                                  d_partial_mins, d_partial_sum_sqs, size, partial_size);
    CHECK(hipDeviceSynchronize());

    double* partial_sums = (double*)malloc(partial_size * sizeof(double));
    double* partial_maxs = (double*)malloc(partial_size * sizeof(double));
    double* partial_mins = (double*)malloc(partial_size * sizeof(double));
    double* partial_sum_sqs = (double*)malloc(partial_size * sizeof(double));
    
    CHECK(hipMemcpy(partial_sums, d_partial_sums, partial_size * sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(partial_maxs, d_partial_maxs, partial_size * sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(partial_mins, d_partial_mins, partial_size * sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(partial_sum_sqs, d_partial_sum_sqs, partial_size * sizeof(double), hipMemcpyDeviceToHost));

    double total_sum = 0.0;
    double global_max = -DBL_MAX;
    double global_min = DBL_MAX;
    double total_sum_sq = 0.0;

    for (int i = 0; i < partial_size; i++) {
        total_sum += partial_sums[i];
        global_max = fmax(global_max, partial_maxs[i]);
        global_min = fmin(global_min, partial_mins[i]);
        total_sum_sq += partial_sum_sqs[i];
    }

    h_stats[0] = total_sum;
    h_stats[1] = global_max;
    h_stats[2] = global_min;
    h_stats[3] = total_sum_sq;

    free(partial_sums);
    free(partial_maxs);
    free(partial_mins);
    free(partial_sum_sqs);
    CHECK(hipFree(d_partial_sums));
    CHECK(hipFree(d_partial_maxs));
    CHECK(hipFree(d_partial_mins));
    CHECK(hipFree(d_partial_sum_sqs));
}

void print_stats(double mass, double max_val, double min_val, double l2_squared, int step) {
    printf("Passo %d:\n", step);
    printf("  Massa total: %.6f\n", mass);
    printf("  Valor máximo: %.6f\n", max_val);
    printf("  Valor mínimo: %.6f\n", min_val);
    printf("  Norma L2: %.6f\n\n", sqrt(l2_squared));
}

int main() {
    int nt = (int)(T / DT);
    double dx = Lx / (Nx - 1);
    double dy = Ly / (Ny - 1);
    double dz = Lz / (Nz - 1);
    double dx2 = dx * dx;
    double dy2 = dy * dy;
    double dz2 = dz * dz;
    double nu_dt = NU * DT;
    int total_size = Nx * Ny * Nz;

    double *u = (double*)malloc(total_size * sizeof(double));
    double *u_new = (double*)malloc(total_size * sizeof(double));
    double *x = (double*)malloc(Nx * sizeof(double));
    double *y = (double*)malloc(Ny * sizeof(double));
    double *z = (double*)malloc(Nz * sizeof(double));
    int i, j, k, n;

    for (i = 0; i < Nx; i++) x[i] = i * dx;
    for (j = 0; j < Ny; j++) y[j] = j * dy;
    for (k = 0; k < Nz; k++) z[k] = k * dz;

    memset(u, 0, total_size * sizeof(double));
    memset(u_new, 0, total_size * sizeof(double));

    double cx = Lx/2, cy = Ly/2, cz = Lz/2;
    for (i = 0; i < Nx; i++) {
        for (j = 0; j < Ny; j++) {
            for (k = 0; k < Nz; k++) {
                double dist2 = pow(x[i] - cx, 2) + pow(y[j] - cy, 2) + pow(z[k] - cz, 2);
                u[i*Ny*Nz + j*Nz + k] = exp(-dist2 / (2 * SIGMA * SIGMA));
            }
        }
    }

    double *d_u, *d_u_new;
    CHECK(hipMalloc(&d_u, total_size * sizeof(double)));
    CHECK(hipMalloc(&d_u_new, total_size * sizeof(double)));
    
    CHECK(hipMemcpy(d_u, u, total_size * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_u_new, u_new, total_size * sizeof(double), hipMemcpyHostToDevice));

    dim3 block_borders(16, 16);
    int max_dim1 = (Nx > Ny) ? Nx : Ny;
    int max_dim2 = (Ny > Nz) ? Ny : Nz;
    if (Nz > max_dim2) max_dim2 = Nz;
    dim3 grid_borders(
        (max_dim1 + block_borders.x - 1) / block_borders.x,
        (max_dim2 + block_borders.y - 1) / block_borders.y
    );
    
    dim3 block_evolve(8, 8, 4);
    dim3 grid_evolve(
        (Nx - 2 + block_evolve.x - 1) / block_evolve.x,
        (Ny - 2 + block_evolve.y - 1) / block_evolve.y,
        (Nz - 2 + block_evolve.z - 1) / block_evolve.z
    );

    double stats[4];
    compute_stats_gpu(d_u, stats, total_size);
    print_stats(stats[0], stats[1], stats[2], stats[3], 0);

    struct timeval t0, t1;
    gettimeofday(&t0, NULL);

    for (n = 0; n < nt; n++) {
        set_borders_zero<<<grid_borders, block_borders>>>(d_u_new, Nx, Ny, Nz);
        CHECK(hipGetLastError());

        evolve_kernel<<<grid_evolve, block_evolve>>>(d_u, d_u_new, nu_dt, dx2, dy2, dz2, Nx, Ny, Nz);
        CHECK(hipGetLastError());

        double* temp = d_u;
        d_u = d_u_new;
        d_u_new = temp;

        if ((n+1) % (nt/10) == 0 || n == nt-1) {
            compute_stats_gpu(d_u, stats, total_size);
            print_stats(stats[0], stats[1], stats[2], stats[3], n+1);
        }
    }

    gettimeofday(&t1, NULL);

    free(u);
    free(u_new);
    free(x);
    free(y);
    free(z);
    CHECK(hipFree(d_u));
    CHECK(hipFree(d_u_new));

    printf("Simulação 3D concluída (CUDA)\n");
    printf("Tempo de execução: %.6lf segundos\n", elapsed(t0, t1));
    printf("Nx = %d, Ny = %d, Nz = %d\n", Nx, Ny, Nz);
    
    return 0;
}