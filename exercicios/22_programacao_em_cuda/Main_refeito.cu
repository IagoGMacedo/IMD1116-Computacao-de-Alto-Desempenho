
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void atualiza(double *vnew, double *vold, int nx, int ny, int nz, double alpha)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = z * ny * nx + y * nx + x;
    if (x > 0 && x < nx - 1 && y > 0 && y < ny - 1 && z > 0 && z < nz - 1)
    {
        int xm = idx - 1;
        int xp = idx + 1;
        int ym = idx - nx;
        int yp = idx + nx;
        int zm = idx - nx * ny;
        int zp = idx + nx * ny;
        vnew[idx] = vold[idx] + alpha * (vold[xp] + vold[xm] +
                                         vold[yp] + vold[ym] +
                                         vold[zp] + vold[zm] - 6 * vold[idx]);
    }
}

int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // declaração de variáveis
    int nx = 381, ny = 381, nz = 381;
    int nt = 381;
    double alpha = 0.1;

    // Aloca memória na CPU para os dados iniciais (h_vold) e resultados (h_result)
    double *h_vold = (double *)malloc(nx * ny * nz * sizeof(double));
    double *h_result = (double *)malloc(nx * ny * nz * sizeof(double));

    // Inicializa h_vold (exemplo: tudo zero, com um ponto central ativo)
    memset(h_vold, 0, nx * ny * nz * sizeof(double));
    h_vold[(nz / 2) * ny * nx + (ny / 2) * nx + (nx / 2)] = 1.0; // fonte no centro

    // Aloca memória na GPU
    double *d_vold, *d_vnew;
    int size = nx * ny * nz * sizeof(double);

    hipMalloc(&d_vold, size);
    hipMalloc(&d_vnew, size);

    // Copia dados iniciais para a GPU
    hipMemcpy(d_vold, h_vold, size, hipMemcpyHostToDevice);

    // Define bloco 3D
    int bx = 8, by = 8, bz = 8;
    dim3 threads(bx, by, bz);
    dim3 grid((nx + bx - 1) / bx, (ny + by - 1) / by, (nz + bz - 1) / bz);
    for (int t = 0; t < nt; t++)
    {
        atualiza<<<grid, threads>>>(d_vnew, d_vold, nx, ny, nz, alpha);
        double *tmp = d_vold;
        d_vold = d_vnew;
        d_vnew = tmp;
    }
    hipMemcpy(h_result, d_vold, size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Tempo: %f ms\n", ms);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    FILE *file = fopen("resultados_gpu.txt", "w");
    if (file == NULL)
    {
        printf("Erro ao abrir o arquivo!\n");
        return 1;
    }

    for (int i = 0; i < nx * ny * nz; i++)
    {
        fprintf(file, "%e\n", h_result[i]);
    }

    fclose(file); // Fecha o arquivo
    printf("Resultados salvos em 'resultados_gpu.txt'.\n");

    return 0;
}