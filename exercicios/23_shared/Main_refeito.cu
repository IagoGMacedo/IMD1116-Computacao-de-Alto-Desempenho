#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 8
#define HALO 1

__global__ void atualiza(double *vnew, double *vold, int nx, int ny, int nz, double alpha)
{
    // Coordenadas do bloco sem halo
    int bx = blockIdx.x * BLOCK_SIZE;
    int by = blockIdx.y * BLOCK_SIZE;
    int bz = blockIdx.z * BLOCK_SIZE;
    
    // Coordenadas locais da thread
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    
    // Memória compartilhada com halo (BLOCK_SIZE + 2*HALO)
    __shared__ double s_data[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2];
    
    // Coordenadas globais para carregamento
    int gx = bx + tx - HALO;
    int gy = by + ty - HALO;
    int gz = bz + tz - HALO;
    
    // Carregar bloco central + halo na memória compartilhada
    if (gx >= 0 && gx < nx && gy >= 0 && gy < ny && gz >= 0 && gz < nz) {
        s_data[tz][ty][tx] = vold[gz * ny * nx + gy * nx + gx];
    } else {
        s_data[tz][ty][tx] = 0.0; // Condições de contorno implícitas
    }
    
    __syncthreads();
    
    // Apenas threads centrais calculam (não fazem parte do halo)
    if (tx >= HALO && tx < BLOCK_SIZE+HALO && 
        ty >= HALO && ty < BLOCK_SIZE+HALO && 
        tz >= HALO && tz < BLOCK_SIZE+HALO) {
        
        // Coordenadas globais para escrita
        gx = bx + tx - HALO;
        gy = by + ty - HALO;
        gz = bz + tz - HALO;
        
        if (gx > 0 && gx < nx-1 && gy > 0 && gy < ny-1 && gz > 0 && gz < nz-1) {
            double val = s_data[tz][ty][tx];
            double sum = s_data[tz][ty][tx+1] + s_data[tz][ty][tx-1] +
                         s_data[tz][ty+1][tx] + s_data[tz][ty-1][tx] +
                         s_data[tz+1][ty][tx] + s_data[tz-1][ty][tx];
            
            vnew[gz * ny * nx + gy * nx + gx] = val + alpha * (sum - 6.0 * val);
        }
    }
}

int main()
{
    // Configuração do problema
    const int nx = 381, ny = 381, nz = 381;
    const int nt = 381;
    const double alpha = 0.1;
    
    // Alocar e inicializar memória na CPU
    double *h_vold = (double*)malloc(nx * ny * nz * sizeof(double));
    double *h_result = (double*)malloc(nx * ny * nz * sizeof(double));
    memset(h_vold, 0, nx * ny * nz * sizeof(double));
    h_vold[(nz/2)*ny*nx + (ny/2)*nx + (nx/2)] = 1.0; // Fonte no centro
    
    // Alocar memória na GPU
    double *d_vold, *d_vnew;
    size_t size = nx * ny * nz * sizeof(double);
    hipMalloc(&d_vold, size);
    hipMalloc(&d_vnew, size);
    hipMemcpy(d_vold, h_vold, size, hipMemcpyHostToDevice);
    
    // Configurar kernel - blocos de 10x10x10 threads (8x8x8 úteis + halo)
    dim3 threads(BLOCK_SIZE+2, BLOCK_SIZE+2, BLOCK_SIZE+2);
    dim3 grid(
        (nx + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (ny + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (nz + BLOCK_SIZE - 1) / BLOCK_SIZE
    );
    
    // Medir tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Executar iterações
    for (int t = 0; t < nt; t++) {
        atualiza<<<grid, threads>>>(d_vnew, d_vold, nx, ny, nz, alpha);
        std::swap(d_vnew, d_vold);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Tempo de execução: %.2f ms\n", ms);
    
    // Copiar resultados e limpar
    hipMemcpy(h_result, d_vold, size, hipMemcpyDeviceToHost);
    
    // Salvar resultados (opcional)
    FILE *file = fopen("resultados_otimizados.txt", "w");
    if (file) {
        for (int i = 0; i < nx*ny*nz; i++) {
            fprintf(file, "%.6e\n", h_result[i]);
        }
        fclose(file);
    }
    
    // Liberar recursos
    hipFree(d_vold);
    hipFree(d_vnew);
    free(h_vold);
    free(h_result);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}